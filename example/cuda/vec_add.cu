#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread is within array bounds
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    // Vector size
    int n = 1000000;
    size_t size = n * sizeof(float);

    // Host vectors
    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device vectors
    float *d_a, *d_b, *d_c;
    
    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Set up execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify results (check first few elements)
    for (int i = 0; i < 5; i++) {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
